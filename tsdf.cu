#include "tsdf.h"

std::vector<Scene3D*> scenes;
std::vector<int> box_id;
int totalObjectCount = 0;
float scale = 100;
float context_pad =3;
std::vector<int> grid_size {3,208,208,100};
int encode_type =100;
int totalScenes = 0;
string file_list = "boxes_NYU_po_test_nb2000_fb.list";

int main(int argc, char **argv){

	int requestedScene = atoi(argv[1]);
	

	FILE* fp = NULL;
	cout << "Loading file: " << file_list << endl << endl;
	fp = fopen(file_list.c_str(),"rb");
	if (fp==NULL) { cout << "Failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }


	while (feof(fp)==0)
	{
		Scene3D* scene = new Scene3D();
		unsigned int len = 0;
		fread((void*)(&len), sizeof(unsigned int), 1, fp);    
		if (len==0) return -1;
		scene->filename.resize(len);
		if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);

		
		string s = scene->filename;
		scene->filename = scene->filename+".bin";

		fread((void*)(scene->R), sizeof(float), 9, fp);
		fread((void*)(scene->K), sizeof(float), 9, fp);
		fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
		fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 


		fread((void*)(&len),    sizeof(unsigned int),   1, fp);
		scene->objects.resize(len);
		if (len>0){
		  totalObjectCount += len;
		  for (int i=0; i<len; ++i){
		      Box3D box;
		      fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
		      fread((void*)(box.base),        sizeof(float), 9, fp);
		      fread((void*)(box.center),      sizeof(float), 3, fp);
		      fread((void*)(box.coeff),       sizeof(float), 3, fp);
		      //process box pad contex oreintation 
		      box = processbox (box, context_pad, grid_size[1]);
		      scene->objects[i]=box;
		      box_id.push_back(i);
		  }
		}
		scenes.push_back(scene);
		totalScenes++;

		if (totalScenes != requestedScene)
		{
			scenes.clear();
			box_id.clear();
			delete scene;
			continue;
		}

		cout << "Scene: " << totalScenes << " Boxes: " << len << " Bin: " << scene->filename << endl << endl;

		//Output files
		FILE* tempname = fopen("temp.txt", "w");
		fprintf(tempname, "%s", s.substr(20).c_str());
		fclose(tempname);
		string tsdffile = "temp.tdsf";

		

		
		float* dataCPUmem = new float[len*3*208*208*100];
		StorageT* dataGPUmem;
		checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (len)*3*208*208*100*sizeof(float)));
	

		
		compute_TSDF(&scenes, &box_id, dataGPUmem,grid_size,encode_type,scale);
		

		
		checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(len)*3*208*208*100*sizeof(float), hipMemcpyDeviceToHost) );
	
		

	
		FILE * fid = fopen(tsdffile.c_str(),"wb");
		fwrite(dataCPUmem,sizeof(float),len*3*208*208*100,fid);
		fclose(fid);
		
		

		//clear for workaround
		scenes.clear();
		box_id.clear();

		//free memory
		delete scene;
		delete[] dataCPUmem;
		hipFree(dataGPUmem);
		
		//Dont calculate others
		break;
	}

	
	fclose(fp);
	return 0;
}


/* USED THIS CODE TO EXTRACT DATA ALREADY
void convertBoxesList()
{
	string box2d = "boxes2d_NYU_po_nb2000.list";
    	
    	FILE* fp2d = fopen(box2d.c_str(),"rb");
    	if (fp2d==NULL) { cout << "Failed to open file: "<< box2d<< endl; exit(EXIT_FAILURE); }


    	while (feof(fp2d)==0) {
      		Scene3D* scene = new Scene3D();
      		unsigned int len = 0;
      		size_t file_size = 0;
      		file_size += fread((void*)(&len), sizeof(unsigned int), 1, fp2d);    
      		if (len==0) break;
      		scene->filename.resize(len);
      		if (len>0) file_size += fread((void*)(scene->filename.data()), sizeof(char), len, fp2d);
     
		int inx = scene->filename.find_last_of("/");
		string output="Boxes//"+scene->filename.substr(inx+1)+".txt";
		FILE* myfile = fopen(output.c_str(), "w");
	

	      	file_size += fread((void*)(scene->R), sizeof(float), 9, fp2d);
		file_size += fread((void*)(scene->K), sizeof(float), 9, fp2d);
		file_size += fread((void*)(&scene->height), sizeof(unsigned int), 1, fp2d);
		file_size += fread((void*)(&scene->width), sizeof(unsigned int), 1, fp2d); 
		file_size += fread((void*)(&len),    sizeof(unsigned int),   1, fp2d);
		scene->objects.resize(len);
      
      		
      		for (int bid = 0;bid<len;bid++){
			//struct Box2D{
			  //unsigned int category;
			  //float tblr[4];
			//};
			Box2D box;
			file_size += fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp2d);
			file_size += fread((void*)(box.tblr),        sizeof(float), 4, fp2d);
			scene->objects_2d_tight.push_back(box);
			 

			fprintf(myfile, "%d %f %f %f %f\n", box.category, box.tblr[0], box.tblr[1], box.tblr[2], box.tblr[3]);


		 	uint8_t hasTarget = 0;
			file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp2d);
			if (hasTarget>0){ cout<<" sth wrong in line "   << __LINE__ << std::endl; }

			file_size += fread((void*)(box.tblr),   sizeof(float), 4, fp2d);
			scene->objects_2d_full.push_back(box);
			file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp2d);
			if (hasTarget>0){ cout<<" sth wrong in line "  << __LINE__ << std::endl; }
      		}
		delete scene;
		fclose(myfile);
    	}
    	fclose(fp2d);
}
*/
